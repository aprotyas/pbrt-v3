#include "hip/hip_runtime.h"

/*
    pbrt source code is Copyright(c) 1998-2016
                        Matt Pharr, Greg Humphreys, and Wenzel Jakob.

    This file is part of pbrt.

    Redistribution and use in source and binary forms, with or without
    modification, are permitted provided that the following conditions are
    met:

    - Redistributions of source code must retain the above copyright
      notice, this list of conditions and the following disclaimer.

    - Redistributions in binary form must reproduce the above copyright
      notice, this list of conditions and the following disclaimer in the
      documentation and/or other materials provided with the distribution.

    THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS
    IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED
    TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
    PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
    HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
    SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
    LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
    DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
    THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
    (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
    OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

 */

// core/integrator.cpp*
#include "integrator.cuh"
#include "scene.cuh"
#include "interaction.cuh"
#include "sampling.cuh"
#include "parallel.cuh"
#include "film.cuh"
#include "sampler.cuh"
#include "integrator.cuh"
#include "progressreporter.cuh"
#include "camera.cuh"
#include "stats.cuh"
#include "utils.cuh"

namespace pbrt {
namespace gpu {

STAT_COUNTER("Integrator/Camera rays traced", nCameraRays);

// Integrator Method Definitions
Integrator::~Integrator() {}

// Integrator Utility Functions
__both__
Spectrum UniformSampleOneLight(const Interaction &it, const Scene &scene,
                               MemoryArena &arena, Sampler &sampler,
                               bool handleMedia, const Distribution1D *lightDistrib) {
    // ProfilePhase p(Prof::DirectLighting);
    // Randomly choose a single light to sample, _light_
    int nLights = utils::get_buffer_size(scene.lights);
    if (nLights == 0) return Spectrum(0.f);
    int lightNum;
    Float lightPdf;
    if (lightDistrib) {
        lightNum = lightDistrib->SampleDiscrete(sampler.Get1D(), &lightPdf);
        if (lightPdf == 0) return Spectrum(0.f);
    } else {
        lightNum = min((int)(sampler.Get1D() * nLights), nLights - 1);
        lightPdf = Float(1) / nLights;
    }
    // TODO: Fix operator::[] call
    const shared_ptr<Light> &light = scene.lights[lightNum];
    Point2f uLight = sampler.Get2D();
    Point2f uScattering = sampler.Get2D();
    // TODO: Fix pointer dereference call
    return EstimateDirect(it, uScattering, *light, uLight,
                          scene, sampler, arena, handleMedia) / lightPdf;
}

__both__
Spectrum EstimateDirect(const Interaction &it, const Point2f &uScattering,
                        const Light &light, const Point2f &uLight,
                        const Scene &scene, Sampler &sampler,
                        MemoryArena &arena, bool handleMedia, bool specular) {
    BxDFType bsdfFlags =
        specular ? BSDF_ALL : BxDFType(BSDF_ALL & ~BSDF_SPECULAR);
    Spectrum Ld(0.f);
    // Sample light source with multiple importance sampling
    Vector3f wi;
    Float lightPdf = 0, scatteringPdf = 0;
    VisibilityTester visibility;
    Spectrum Li = light.Sample_Li(it, uLight, &wi, &lightPdf, &visibility);
    if (lightPdf > 0 && !Li.IsBlack()) {
        // Compute BSDF or phase function's value for light sample
        Spectrum f;
        if (it.IsSurfaceInteraction()) {
            // Evaluate BSDF for light sampling strategy
            const SurfaceInteraction &isect = (const SurfaceInteraction &)it;
            f = isect.bsdf->f(isect.wo, wi, bsdfFlags) *
                AbsDot(wi, isect.shading.n);
            scatteringPdf = isect.bsdf->Pdf(isect.wo, wi, bsdfFlags);
        }
        if (!f.IsBlack()) {
            // Compute effect of visibility for light source sample
              if (!visibility.Unoccluded(scene)) {
                Li = Spectrum(0.f);
            }

            // Add light's contribution to reflected radiance
            if (!Li.IsBlack()) {
                if (IsDeltaLight(light.flags))
                    Ld += f * Li / lightPdf;
                else {
                    Float weight =
                        PowerHeuristic(1, lightPdf, 1, scatteringPdf);
                    Ld += f * Li * weight / lightPdf;
                }
            }
        }
    }

    // Sample BSDF with multiple importance sampling
    if (!IsDeltaLight(light.flags)) {
        Spectrum f;
        bool sampledSpecular = false;
        if (it.IsSurfaceInteraction()) {
            // Sample scattered direction for surface interactions
            BxDFType sampledType;
            const SurfaceInteraction &isect = (const SurfaceInteraction &)it;
            f = isect.bsdf->Sample_f(isect.wo, &wi, uScattering, &scatteringPdf,
                                     bsdfFlags, &sampledType);
            f *= AbsDot(wi, isect.shading.n);
            sampledSpecular = (sampledType & BSDF_SPECULAR) != 0;
        }
        if (!f.IsBlack() && scatteringPdf > 0) {
            // Account for light contributions along sampled direction _wi_
            Float weight = 1;
            if (!sampledSpecular) {
                lightPdf = light.Pdf_Li(it, wi);
                if (lightPdf == 0) return Ld;
                weight = PowerHeuristic(1, scatteringPdf, 1, lightPdf);
            }

            // Find intersection and compute transmittance
            SurfaceInteraction lightIsect;
            Ray ray = it.SpawnRay(wi);
            Spectrum Tr(1.f);
            bool foundSurfaceInteraction =
                handleMedia ? scene.IntersectTr(ray, sampler, &lightIsect, &Tr)
                            : scene.Intersect(ray, &lightIsect);

            // Add light contribution from material sampling
            Spectrum Li(0.f);
            if (foundSurfaceInteraction) {
                if (lightIsect.primitive->GetAreaLight() == &light)
                    Li = lightIsect.Le(-wi);
            } else
                Li = light.Le(ray);
            if (!Li.IsBlack()) Ld += f * Li * Tr * weight / scatteringPdf;
        }
    }
    return Ld;
}

// SamplerIntegrator Method Definitions
void SamplerIntegrator::Render(const Scene &scene) {
    Preprocess(scene, *sampler);
    // Render image tiles in parallel

    // Compute number of tiles, _nTiles_, to use for parallel rendering
    Bounds2i sampleBounds = camera->film->GetSampleBounds();
    Vector2i sampleExtent = sampleBounds.Diagonal();
    const int tileSize = 16;
    Point2i nTiles((sampleExtent.x + tileSize - 1) / tileSize,
                   (sampleExtent.y + tileSize - 1) / tileSize);
    ProgressReporter reporter(nTiles.x * nTiles.y, "Rendering");
    {
        ParallelFor2D([&](Point2i tile) {
            // Render section of image corresponding to _tile_

            // Allocate _MemoryArena_ for tile
            MemoryArena arena;

            // Get sampler instance for tile
            int seed = tile.y * nTiles.x + tile.x;
            std::unique_ptr<Sampler> tileSampler = sampler->Clone(seed);

            // Compute sample bounds for tile
            int x0 = sampleBounds.pMin.x + tile.x * tileSize;
            int x1 = min(x0 + tileSize, sampleBounds.pMax.x);
            int y0 = sampleBounds.pMin.y + tile.y * tileSize;
            int y1 = min(y0 + tileSize, sampleBounds.pMax.y);
            Bounds2i tileBounds(Point2i(x0, y0), Point2i(x1, y1));

            // Get _FilmTile_ for tile
            std::unique_ptr<FilmTile> filmTile =
                camera->film->GetFilmTile(tileBounds);

            // Loop over pixels in tile to render them
            for (Point2i pixel : tileBounds) {
                {
                    ProfilePhase pp(Prof::StartPixel);
                    tileSampler->StartPixel(pixel);
                }

                // Do this check after the StartPixel() call; this keeps
                // the usage of RNG values from (most) Samplers that use
                // RNGs consistent, which improves reproducability /
                // debugging.
                if (!InsideExclusive(pixel, pixelBounds))
                    continue;

                do {
                    // Initialize _CameraSample_ for current sample
                    CameraSample cameraSample =
                        tileSampler->GetCameraSample(pixel);

                    // Generate camera ray for current sample
                    RayDifferential ray;
                    Float rayWeight =
                        camera->GenerateRayDifferential(cameraSample, &ray);
                    ray.ScaleDifferentials(
                        1 / std::sqrt((Float)tileSampler->samplesPerPixel));
                    ++nCameraRays;

                    // Evaluate radiance along camera ray
                    Spectrum L(0.f);
                    // if (rayWeight > 0) L = Li(ray, scene, *tileSampler, arena);
                    if (rayWeight > 0) CallLi<<<1,1>>>(ray, scene, *tileSampler, arena, 0, *this);

                    // Issue warning if unexpected radiance value returned
                    if (L.HasNaNs()) {
                        L = Spectrum(0.f);
                    } else if (L.y() < -1e-5) {
                        L = Spectrum(0.f);
                    } else if (pbrt::gpu::isinf(L.y())) {
                        L = Spectrum(0.f);
                    }

                    // Add camera ray's contribution to image
                    filmTile->AddSample(cameraSample.pFilm, L, rayWeight);

                    // Free _MemoryArena_ memory from computing image sample
                    // value
                    arena.Reset();
                } while (tileSampler->StartNextSample());
            }

            // Merge image tile into _Film_
            camera->film->MergeFilmTile(std::move(filmTile));
            reporter.Update();
        }, nTiles);
        reporter.Done();
    }

    // Save final image after rendering
    camera->film->WriteImage();
}

__global__
void CallLi(const RayDifferential &ray, const Scene &scene,
                        Sampler &sampler, MemoryArena &arena,
                        int depth, SamplerIntegrator &integrator) {
    integrator.Li(ray, scene, sampler, arena, depth);
}

}  // namespace gpu
}  // namespace pbrt
